#include "hip/hip_runtime.h"
#include "c-product.cuh"
#include <stdio.h>

__global__
void product_iterator(unsigned char* set[], int size[], int count, long int all)
{
    int *index;
    index = (int *)malloc(count * sizeof(int));

    // Block and thread params
    long int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // init the index list
    long int num = init;
    updateIndex<<<1, 1>>>(index, size, count, num, all);

    for (int i = 0; i < all; i++) {
        // process the subkeys
        for (int j = 0;j < count; j++){
            // do something;
        }
        // update the index according to the blockDim.x and gridDim.x
        num += stride;
        updateIndex<<<1, 1>>>(index, size, count, num, all);

        // update the index list
        i = all;
    }

    free(index);
    return;
}

__global__
void updateIndex(int index[], int size[], int count, long int num, long int all){
    long int weight = all;
    for (int i = 0; i < count; i++){
        weight /= size[i];
        index[i] = num / weight;
        num -= index[i] * weight;
    }
}