#include "hip/hip_runtime.h"
#include "c-product.cuh"
#include <stdio.h>
#include <stdlib.h>

__global__
void product_iterator(unsigned char* set[], int size[], int count, long int all)
{
    int *index;
    index = (int *)malloc(count * sizeof(int));

    int *delta;
    delta = (int *)malloc(count * sizeof(int));

    // Block and thread params
    long int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    updateIndex(delta, size, count, stride, all);
    updateIndex(index, size, count, init, all);

    for (long int i = init; i < all; i+= stride) {
        // update the index according to the blockDim.x and gridDim.x
        // process the subkeys
        for (int j = 0;j < count; j++){
            set[j][index[j]] += index[j];
        }
        // printf("\n");
        // printf("%d", i);
        if (i + stride < all){
            increment(index, size, count, delta);
        }
    }

    free(index);
    return;
}

__device__
void updateIndex(int index[], int size[], int count, long int num, long int all){
    long int weight = all;
    for (int i = 0; i < count; i++){
        weight /= size[i];
        index[i] = num / weight;
        num -= index[i] * weight;
    }
}

__device__
void increment(int index[], int size[], int count, int delta[]){
    for (int i = count-1; i >= 0; i--){
        index[i] += delta[i];
        while(index[i] >= size[i]){
            index[i] -= size[i];
            index[i-1]++;
        }
    }
}