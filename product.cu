#include "c-product.cuh"
#include <stdio.h>

#define THREADS 256
#define BLOCKS 16

int main()
{
    int count = 15;
    int elements_count = 4;

    // The sets' pointer
    unsigned char **set;
    hipMallocManaged(&set, count * sizeof(unsigned char *));

    // // The real sets
    int* size;
    hipMallocManaged(&size, count * sizeof(int));

    for (int i = 0;i < count; i++){
        size[i] = elements_count;
        hipMallocManaged(&(set[i]), size[i] * sizeof(unsigned char));
        for (int j = 0; j < size[i]; j++){
            set[i][j] = 0x11 * j;
        }
    }

    long int all = 1;
    for (int i = 0; i < count; i++) {
        all *= size[i];
    }

    printf("%ld\n", all);

    product_iterator<<<BLOCKS, THREADS>>>(set, size, count, all);
    hipDeviceSynchronize();

    // Free memory
    for (int i = 0;i < count; i ++){
        hipFree(set[i]);
    }
    hipFree(set);
}